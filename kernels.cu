#include "hip/hip_runtime.h"
#include "kernels.h"

#include <ctime>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "hip/hip_runtime_api.h"

/***** CUDA Kernels *****/

__global__ void
setupRandomVectorGen(hiprandState* state, unsigned long seed, unsigned int numElems) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= numElems) return;
    hiprand_init(seed, id, 0, &(state[id]));
}

__global__ void
runRandomVectorGen(float* vec, hiprandState* globalState, float threshold, unsigned int numElems) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= numElems) return;
    hiprandState localState = globalState[id];
    float rndVal = hiprand_uniform(&localState);
    
    vec[id] = (rndVal * 2 * threshold) - threshold;
}

__global__ void
updateParams(float* params, float* derivatives, float* weights, float learnRate, unsigned int numElems) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= numElems) return;
    float epsilon = 0.0001;
    weights[id] += derivatives[id]*derivatives[id];
    params[id] -= (learnRate * derivatives[id])/(sqrt(weights[id]) + epsilon);
}

/***** Kernel Wrappers *****/

void
kernelRandomwordVecs(ParamMem_t& params, float threshold) {
    timeval tim;
    gettimeofday(&tim, NULL);
    double t1=tim.tv_sec+(tim.tv_usec/1000000.0);

    unsigned int blockSize = 1024;
    unsigned int numElems = params.nWords * params.wordVecDim;
    unsigned int numBlocks = numElems / blockSize + 1;
    dim3 threadsPerBlock(blockSize, 1, 1);
    hiprandState* devState;
    checkCudaErrors(hipMalloc((void**)&devState, numElems*sizeof(hiprandState)));
    setupRandomVectorGen<<<numBlocks, threadsPerBlock>>>(devState, time(NULL), numElems);
    runRandomVectorGen<<<numBlocks, threadsPerBlock>>>(params.wordVecs, devState, threshold, numElems);
    
    checkCudaErrors(hipFree(devState));

    gettimeofday(&tim, NULL);
    double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
    printf("Random word vectors time: %f\n", t2-t1);
}

void
kernelUpdateParams(ParamMem_t& params, ParamMem_t& derivatives,
  ParamMem_t& adagradWeights, float learnRate) {
    timeval tim;
    gettimeofday(&tim, NULL);
    double t1=tim.tv_sec+(tim.tv_usec/1000000.0);


    unsigned int blockSize = 1024;
    unsigned int numElems = params.totalSize;
    unsigned int numBlocks = numElems / blockSize + 1;
    dim3 threadsPerBlock(blockSize, 1, 1);
    updateParams<<<numBlocks,threadsPerBlock>>>(params.base, derivatives.base,
        adagradWeights.base, learnRate, params.totalSize);
    //checkCudaErrors(hipGetLastError());

    gettimeofday(&tim, NULL);
    double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
    printf("Update params time: %f\n", t2-t1);
}